#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

#define NAMELEN 255

#define CSC(call) do { 		\
	hipError_t e = call;	\
	if (e != hipSuccess) {	\
		fprintf(stderr, "ERROR: %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
		exit(0);			\
	}						\
} while(0)




texture<uchar4, 2, hipReadModeElementType> tex;


__global__ void kernel(uchar4 *dev_data, int w, int h, int r, float *dev_a, bool isColumns){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int offsetx = gridDim.x * blockDim.x;
	int offsety = gridDim.y * blockDim.y;


	uchar4 pixel;
	float red, green, blue;

	for(int x = idx; x < w; x += offsetx){
		for(int y = idy; y < h; y += offsety){
			red = 0.0;
			green = 0.0;
			blue = 0.0;

			for(int i = -r; i <=r; i++){

				if (isColumns) {
				pixel = tex2D(tex, x + i, y);
				}
				else {
				pixel = tex2D(tex, x, y + i);
				}

				red += dev_a[r + i] * pixel.x;
				green += dev_a[r + i] * pixel.y;
				blue += dev_a[r + i] * pixel.z;
			}

			dev_data[y * w + x] = make_uchar4(red, green, blue, 0.0);
		}
	}
}

class CUGaussianBlur {
public:

	float *dev_a;
	hipArray *arr;
	uchar4 *data;
	uchar4 *dev_data;
	hipChannelFormatDesc ch;

	int w;
	int h;

	CUGaussianBlur(int r, int _w, int _h, uchar4 *_data)  {

		w = _w;
		h = _h;
		data = _data;

		
		int n = 2 * r + 1;
		float sum = 0.0;

		float a[n];

		for(int i = -r; i <= r; i++) {
			a[i + r] = exp(-1.0 * (i * i) / (2 * r * r));
			sum += a[i + r];
		}

		for(int i = 0; i < n; i++){
			a[i] /= sum;
		}

		CSC(hipMalloc(&dev_a, sizeof(float) * n));
		CSC(hipMemcpy(dev_a, a, sizeof(float) * n, hipMemcpyHostToDevice));

		
		ch = hipCreateChannelDesc<uchar4>();
		CSC(hipMallocArray(&arr, &ch, w, h));
		CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));

		tex.addressMode[0] = hipAddressModeClamp;
		tex.addressMode[1] = hipAddressModeClamp;
		tex.channelDesc = ch;
		tex.filterMode = hipFilterModePoint;
		tex.normalized = false;

		CSC(hipBindTextureToArray(tex, arr, ch));
		CSC(hipMalloc(&dev_data, sizeof(uchar4) * w * h));

	} 

	void run_kernel(int r) {
		kernel <<<dim3(8,16), dim3(8,32) >>> (dev_data, w, h, r, dev_a, false);

		CSC(hipUnbindTexture(tex));
		CSC(hipMemcpyToArray(arr, 0, 0, dev_data, sizeof(uchar4) * w * h, hipMemcpyDeviceToDevice));
		CSC(hipBindTextureToArray(tex, arr, ch));

		kernel <<< dim3(8,16), dim3(16,32) >>> (dev_data, w, h, r, dev_a, true);


		CSC(hipMemcpy(data, dev_data, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	}

	~CUGaussianBlur(){

		free(data);

		CSC(hipUnbindTexture(tex));
		CSC(hipFreeArray(arr));
		CSC(hipFree(dev_data));
		CSC(hipFree(dev_a));	
	}
};


int main() {
	int r, w, h;
	
	char input_name[256];
	char output_name[256];


	scanf("%s", input_name);
	scanf("%s", output_name);
	scanf("%d", &r);

	FILE *in = fopen(input_name, "rb");

	fread(&w, sizeof(int), 1, in);
	fread(&h, sizeof(int), 1, in);

	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	

	fread(data, sizeof(uchar4), w * h, in);
	fclose(in);

	CUGaussianBlur image = CUGaussianBlur(r, w, h, data);
	if(r != 0){
		
		image.run_kernel(r);
	}

	FILE *out = fopen(output_name, "wb");
	fwrite(&w, sizeof(int), 1, out);
	fwrite(&h, sizeof(int), 1, out);
	fwrite(image.data, sizeof(uchar4), w * h, out);

	fclose(out);

	return 0;
}